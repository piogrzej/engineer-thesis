#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>

#include "Logger.h"

bool initCuda(int argc, char **argv)
{

    hipError_t cudaStatus;
    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        return false;
    }
    int cuda_device = findCudaDevice(argc, (const char **)argv);
        hipDeviceProp_t deviceProps;
        checkCudaErrors(hipGetDeviceProperties(&deviceProps, cuda_device));

    int cdpCapable = (deviceProps.major == 3 && deviceProps.minor >= 5) || deviceProps.major >=4;

    printf("GPU: %s (SM %d.%d)\n", deviceProps.name, deviceProps.major, deviceProps.minor);

    if (!cdpCapable)
    {
        ErrorLogger::getInstance() >> "RandomWalk potrzebuje SM 3.5 lub wyzszej do CUDA Dynamic Parallelism.\n";
        return false;
    }
    return true;
}
