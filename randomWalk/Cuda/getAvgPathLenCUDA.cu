#include "mainkernels.h"
#include "hip/hip_runtime_api.h"
#include "../utils/Timer.h"

floatingPoint countAvg(unsigned int output[],int ITER_NUM)
{
    floatingPoint out=0;

    for(unsigned int i=0; i<ITER_NUM;++i)
    {
        out += output[i];
    }

    return out/ITER_NUM;
}

floatingPoint getAvgPathLenCUDA(char* path, int ITER_NUM,int RECT_ID,bool measure)
{
    //tworzenie drzewa
    QuadTreeManager* qtm = randomWalkCudaInit(path,measure);
    //alokowanie pamieci na wynik
    unsigned int output[ITER_NUM];
    unsigned int* d_output;
    printf("Test: %s watkow: %d\n", path,ITER_NUM);
    unsigned int outputSize = ITER_NUM * sizeof(unsigned int);
    if(true==measure)
	{
		Timer::getInstance().start("_RandomWalkCuda Total");
	}
    checkCudaErrors(hipMalloc((void **)&d_output,outputSize));
    randomWalkCudaWrapper(1,ITER_NUM,qtm,RECT_ID,d_output,time(NULL));
    checkCudaErrors(hipMemcpy(output,d_output,outputSize,hipMemcpyDeviceToHost));
    if(true==measure)
	{
		Timer::getInstance().stop("_RandomWalkCuda Total");
	}
    freeQuadTreeManager(qtm);
    hipFree(d_output);
    hipDeviceReset();
    return countAvg(output,ITER_NUM);
}
