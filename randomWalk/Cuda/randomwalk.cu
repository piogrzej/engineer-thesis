#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <vector>

#include "createquadtree.h"
#include "../Parser.h"
#include "../Logger.h"
#include "../Timer.h"
// TO DO: brzydkie kopiowanie, trzeba poprawić
// TO DO: wykrywanie ilosci threadow, thread/block, (hipDeviceProp_t)
void randomWalkCUDA(char* path, int ITER_NUM, int RECT_ID)
{
    ErrorLogger::getInstance() >> "Random Walk CUDA\n";
    Timer::getInstance().start("Parser");
    Parser parser(path, "<<");
    const std::vector<RectHost>& layer = parser.getLayerAt(0); // na razie 0 warstwa hardcode
    RectHost const& spaceSize = parser.getLayerSize(0);
    Timer::getInstance().stop("Parser");
    QuadTreeManager treeMng = createQuadTree(layer,spaceSize,false);


    Timer::getInstance().printResults();
}
