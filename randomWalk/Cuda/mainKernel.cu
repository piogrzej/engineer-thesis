
#include "hip/hip_runtime.h"
#include ""
#include "../Parser.h"

#include <hip/hip_runtime_api.h>
#include <stdio.h>


bool initCuda(int argc, char **argv)
{
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        return false;
    }
    int cuda_device = findCudaDevice(argc, (const char **)argv);
        hipDeviceProp_t deviceProps;
        checkCudaErrors(hipGetDeviceProperties(&deviceProps, cuda_device));
    int cdpCapable = (deviceProps.major == 3 && deviceProps.minor >= 5) || deviceProps.major >=4;

    printf("GPU: %s ma (SM %d.%d)\n", deviceProps.name, deviceProps.major, deviceProps.minor);

    if (!cdpCapable)
    {
        std::cerr << "RandomWalk potrzebuje SM 3.5 lub wyzszej do CUDA Dynamic Parallelism.\n" << std::endl;
        return false;
    }
    return true;
}


void randomWalkCUDA(char* path, int ITER_NUM, int RECT_ID)
{
    Parser parser(path, "<<");
    const Layer layer = parser.getLayerAt(0);
    Rect const& spaceSize = parser.getLayerSize(0);
}
