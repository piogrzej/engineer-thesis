#include "mainkernels.h"

void freeQuadTreeManager(QuadTreeManager* qtm)
{
	QuadTreeManager *hostQTM= new QuadTreeManager();
	hipMemcpy(hostQTM,qtm,sizeof(QuadTreeManager),hipMemcpyDeviceToHost);
    hipFree(hostQTM->d_intg);
    hipFree(hostQTM->nodes);
    hipFree(hostQTM->rects);
    hipFree(hostQTM->root);
    hipFree(qtm);
}
