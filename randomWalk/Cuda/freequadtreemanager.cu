#include "mainkernels.h"

void freeQuadTreeManager(QuadTreeManager* qtm)
{
    hipFree(qtm->d_intg);
    hipFree(qtm->nodes);
    hipFree(qtm->rects);
    hipFree(qtm->root);
    hipFree(qtm);
}
