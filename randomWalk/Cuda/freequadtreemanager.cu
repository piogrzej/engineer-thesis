#include "mainkernels.h"

void freeQuadTreeManager(QuadTreeManager* qtm)
{
	QuadTreeManager hostQTM;
	hipMemcpy(&hostQTM,qtm,sizeof(QuadTreeManager),hipMemcpyDeviceToHost);
    hipFree(hostQTM.d_intg);
    hipFree(hostQTM.nodes);
    hipFree(hostQTM.rects);
    hipFree(qtm);
}
